#include "hip/hip_runtime.h"
﻿// Created by Aslı Başak CİVEK by modifying the Ascon CUDA codes of Cihangir TEZCAN: https://github.com/cihangirtezcan/CUDA_ASCON
// Windows version

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <inttypes.h>
#include <math.h>
#include "hip/hip_runtime.h"
#include ""
#include <Windows.h>
#include "rdrand.h"
#include <intrin.h>
#include <immintrin.h>
#include <string.h>

#define bit64 unsigned __int64
#define RDRAND_MASK 0x40000000
#define RETRY_LIMIT 10


// for random number generation //
int RdRand_cpuid() {
	int info[4] = { -1, -1, -1, -1 };
	__cpuid(info, 0);
	if (memcmp((void*)&info[1], (void*)"Genu", 4) != 0 ||
		memcmp((void*)&info[3], (void*)"ineI", 4) != 0 ||
		memcmp((void*)&info[2], (void*)"ntel", 4) != 0) {
		return 0;
	}
	__cpuid(info, 1);
	int ecx = info[2];
	if ((ecx & RDRAND_MASK) == RDRAND_MASK)
		return 1;
	else
		return 0;
}

int RdRand_isSupported() {
	static int supported = RDRAND_SUPPORT_UNKNOWN;
	if (supported == RDRAND_SUPPORT_UNKNOWN) {
		if (RdRand_cpuid())
			supported = RDRAND_SUPPORTED;
		else
			supported = RDRAND_UNSUPPORTED;
	}
	return (supported == RDRAND_SUPPORTED) ? 1 : 0;
}

int rdrand_64(uint64_t* x, int retry) {
	if (RdRand_isSupported()) {
		if (retry) {
			for (int i = 0; i < RETRY_LIMIT; i++) {
				if (_rdrand64_step(x))
					return RDRAND_SUCCESS;
			}
			return RDRAND_NOT_READY;
		}
		else {
			if (_rdrand64_step(x))
				return RDRAND_SUCCESS;
			else
				return RDRAND_NOT_READY;
		}
	}
	else {
		return RDRAND_UNSUPPORTED;
	}
}

// for random number generation //

double PCFreq = 0.0;
__int64 CounterStart = 0;
void StartCounter() {
	LARGE_INTEGER li;
	if (!QueryPerformanceFrequency(&li))
		printf("QueryPerformanceFrequency failed!\n");

	PCFreq = double(li.QuadPart) / 1000.0;

	QueryPerformanceCounter(&li);
	CounterStart = li.QuadPart;
}

// #######################  SYCON EXPERIMENT - CPU CODE ######################
bit64 t[7];
bit64 state[5] = { 0x00000000000000001,0x0000000000000000, 0x0000000000000000, 0x0000000000000000, 0x0000000000000000 };
bit64 state2[5] = { 0 }; // pair
// to verify the test vectors:
bit64 test_state[5] = { 0x0000000000000000, 0x0000000000000000, 0x00000000aaaaaaaa, 0xaaaaaaaaaaaaaaaa, 0xaaaaaaaaaaaaaaaa };
// sycon constant
bit64 constant[12] = { 0x05aaaaaaaaaaaaaa,0x0aaaaaaaaaaaaaaa,0x0daaaaaaaaaaaaaa,0x0eaaaaaaaaaaaaaa,0x0faaaaaaaaaaaaaa,0x07aaaaaaaaaaaaaa,0x03aaaaaaaaaaaaaa,0x01aaaaaaaaaaaaaa,0x08aaaaaaaaaaaaaa,0x04aaaaaaaaaaaaaa,0x02aaaaaaaaaaaaaa,0x09aaaaaaaaaaaaaa };

void print_state(bit64 state[5]) {
	int i;
	for (i = 0; i < 5; i++) printf("%016llx\n", state[i]);
	printf("\n");
}

// SYCON SBOX
void substitution(bit64 x[5]) { // x0 as lsb
	t[0] = x[2] ^ x[4]; t[1] = t[0] ^ x[1]; t[2] = x[1] ^ x[3]; t[3] = x[0] ^ x[4]; t[4] = t[1] & x[3];
	t[5] = t[3] ^ t[4]; x[1] = ~x[1]; x[1] = x[1] & x[3]; t[6] = ~t[2]; t[6] = t[6] & x[0];
	x[1] = x[1] ^ t[1]; x[1] = x[1] ^ t[6]; t[3] = ~t[3]; t[6] = t[3] & x[2]; t[1] = t[6] ^ t[2];
	t[0] = ~t[0]; x[3] = t[0] & x[3]; x[3] = x[3] ^ x[0]; x[2] = ~x[2]; x[3] = x[3] ^ x[2];
	x[4] = ~x[4]; x[4] = x[4] & x[0]; x[4] = x[4] ^ t[2]; x[0] = t[5]; x[2] = t[1];

}

bit64 l_rotate(bit64 x, int l) {
	bit64 temp;
	temp = (x << l) ^ (x >> (64 - l));
	return temp;

}

void sycon_linear_layer(bit64 state[5]) {
	bit64 temp0, temp1, temp2;

	temp0 = l_rotate(state[0], 59);
	temp1 = l_rotate(state[0], 54);
	temp2 = state[0] ^ temp0 ^ temp1;
	state[0] = l_rotate(temp2, 40);

	temp0 = l_rotate(state[1], 55);
	temp1 = l_rotate(state[1], 46);
	temp2 = state[1] ^ temp0 ^ temp1;
	state[1] = l_rotate(temp2, 32);

	temp0 = l_rotate(state[2], 33);
	temp1 = l_rotate(state[2], 2);
	temp2 = state[2] ^ temp0 ^ temp1;
	state[2] = l_rotate(temp2, 16);

	temp0 = l_rotate(state[3], 21);
	temp1 = l_rotate(state[3], 42);
	temp2 = state[3] ^ temp0 ^ temp1;
	state[3] = l_rotate(temp2, 56);

	temp0 = l_rotate(state[4], 13);
	temp1 = l_rotate(state[4], 26);
	state[4] ^= temp0 ^ temp1;

}
void permutation(bit64 state[5], int round) {
	int i;
	for (i = 0; i < round; i++) {
		// SBox (SB), SubBlockDiffusion (SD), AddRound-Const(RC)
		substitution(state);
		sycon_linear_layer(state);
		//add round constant
		//state[2] = state[2] ^ constant[i]; // negligible for the experiment
	}
}

// to verify the test vectors
void test_permutation(bit64 state[5], int round) {
	int i;
	printf("Round: 0 STATE:\n");
	print_state(state);
	for (i = 0; i < round; i++) {
		// SBox (SB), SubBlockDiffusion (SD), AddRound-Const(RC)

		substitution(state);
		printf("Round: %d SBOX:\n", i + 1);
		print_state(state);

		sycon_linear_layer(state);
		printf("Round: %d LINEAR\n", i + 1);
		print_state(state);

		//add round constant 
		state[2] = state[2] ^ constant[i];
		printf("Round: %d ADD CONSTANT\n", i + 1);
		print_state(state);
	}
}

// hex xor calculation
int parity(unsigned long long v) { // 64-bit word 
	int a;
	v ^= v >> 1;
	v ^= v >> 2;
	v = (v & 0x1111111111111111UL) * 0x1111111111111111UL;
	a = (v >> 60) & 1;
	return a; //Parity of Xi
}

// it uses the permutation of the first state as a random state
int experiment(bit64 state[5], int round) {

	//flip some bits to get state2
	state2[0] = state[0];
	state2[1] = state[1] ^ 0x0000000000800000; // should be from the successful experiment
	state2[2] = state[2];
	state2[3] = state[3] ^ 0x0000000000800000; // should be from the successful experiment
	state2[4] = state[4];

	permutation(state, round);
	permutation(state2, round);

	// change it when needed:
	// 2r type-II linear approx - for 4 round experiment:
	//return parity((state[0] ^ state2[0]) & 0x66EEECDDD9BBB377);

	// 2r type-I linear approx - for 4 round experiment:
	//return parity(((state[1] ^ state2[1]) & 0xBB6ED9B76DDB76CD) ^ ((state[3] ^ state2[3]) & 0xFF000007FFFFFFFF) ^ ((state[4] ^ state2[4]) & 0xB6D6DB5B6F6DADB7));
	// approxes: 0x0000000004000000 for x0, x1, x2

	// 3r type-II linear approx - for 5 round experiment:
	//return parity((state[0] ^ state2[0]) & 0xB37766EECDDDD9BB);

	// 3r typeI for 5-round experiment
	return parity(((state[0] ^ state2[0]) & 0x576eaedd55faabb7) ^ ((state[1] ^ state2[1]) & 0x6c1b168d8362d9b4) ^ ((state[3] ^ state2[3]) & 0x001ffffffffff800));
}

bit64 cpu_single_experiment(int round, bit64 experiment_size) {
	bit64 a, s, counter;
	counter = 0;
	bit64 bias;

	// Run the experiment 1024*1024*32*trial times for r round 
	for (s = 0; s < experiment_size; s++) {
		a = experiment(state, round);
		if (a == 0) { counter = counter + 1; }
	}

	bias = (experiment_size) / 2 - counter;
	printf("Size: %lld, Counter: %lld, Bias: %lld, Time: %u seconds\n", s, counter, bias, clock() / CLOCKS_PER_SEC);
	return bias;
}


// #######################  SYCON EXPERIMENT - GPU CODE ######################
#define BLOCKS 32
#define THREADS 1024
#define TRIALS	1024 //*1024
__int64 trial = 1, trial_i = 0, repeat = 1; //should be 10 for calculating the average.
bit64* nonce, * nonce_d;

// device functions for rotation
__device__ bit64 l_rotate_d(bit64 x, int l) {
	bit64 temp;
	temp = (x << l) ^ (x >> (64 - l));
	return temp;

}

__global__ void gpu_single_experiment(bit64 nonce[], __int64 counter[], int round) {
	int threadIndex = blockIdx.x * blockDim.x + threadIdx.x;
	bit64 initial0, initial1, initial2, initial3, initial4;
	bit64 pair0, pair1, pair2, pair3, pair4;
	bit64 t0, t1, t2, t3, t4, t5, t6;

	initial0 = 0x0000000000000000; // key 
	initial1 = 0x0000000000000000; // key
	initial2 = nonce[2 * threadIndex]; // nonce
	initial3 = nonce[2 * threadIndex + 1]; // nonce
	initial4 = 0x0000000000000000; // iv for sycon64-aead

	for (int c = 0; c < TRIALS; c++) {

		// for the 4-round experiment, change it to: 
		/*
		pair0 = initial0 ^ 0x0010000000000000;
		pair1 = initial1;
		pair2 = initial2 ^ 0x0010000000000000;
		pair3 = initial3;
		pair4 = initial4 ^ 0x0010000000000000;
		*/

		// for the 5-round experiment, change it to: 
		pair0 = initial0;
		pair1 = initial1 ^ 0x0000000000800000; // should be from the successful experiment
		pair2 = initial2;
		pair3 = initial3 ^ 0x0000000000800000; // should be from the successful experiment
		pair4 = initial4;

		for (int i = 0; i < round; i++) {

			// Sycon Sbox // initial
			t0 = initial2 ^ initial4; t1 = t0 ^ initial1; t2 = initial1 ^ initial3; t3 = initial0 ^ initial4; t4 = t1 & initial3;
			t5 = t3 ^ t4; initial1 = ~initial1; initial1 = initial1 & initial3; t6 = ~t2; t6 = t6 & initial0;
			initial1 = initial1 ^ t1; initial1 = initial1 ^ t6; t3 = ~t3; t6 = t3 & initial2; t1 = t6 ^ t2;
			t0 = ~t0; initial3 = t0 & initial3; initial3 = initial3 ^ initial0; initial2 = ~initial2; initial3 = initial3 ^ initial2;
			initial4 = ~initial4; initial4 = initial4 & initial0; initial4 = initial4 ^ t2; initial0 = t5; initial2 = t1;

			// Sycon Liner layer // initial
			initial0 = l_rotate_d(initial0 ^ l_rotate_d(initial0, 59) ^ l_rotate_d(initial0, 54), 40);
			initial1 = l_rotate_d(initial1 ^ l_rotate_d(initial1, 55) ^ l_rotate_d(initial1, 46), 32);
			initial2 = l_rotate_d(initial2 ^ l_rotate_d(initial2, 33) ^ l_rotate_d(initial2, 2), 16);
			initial3 = l_rotate_d(initial3 ^ l_rotate_d(initial3, 21) ^ l_rotate_d(initial3, 42), 56);
			initial4 = initial4 ^ l_rotate_d(initial4, 13) ^ l_rotate_d(initial4, 26);
		}

		for (int i = 0; i < round; i++) {
			// Sycon sbox // pair
			t0 = pair2 ^ pair4; t1 = t0 ^ pair1; t2 = pair1 ^ pair3; t3 = pair0 ^ pair4; t4 = t1 & pair3;
			t5 = t3 ^ t4; pair1 = ~pair1; pair1 = pair1 & pair3; t6 = ~t2; t6 = t6 & pair0;
			pair1 = pair1 ^ t1; pair1 = pair1 ^ t6; t3 = ~t3; t6 = t3 & pair2; t1 = t6 ^ t2;
			t0 = ~t0; pair3 = t0 & pair3; pair3 = pair3 ^ pair0; pair2 = ~pair2; pair3 = pair3 ^ pair2;
			pair4 = ~pair4; pair4 = pair4 & pair0; pair4 = pair4 ^ t2; pair0 = t5; pair2 = t1;

			// Liner layer // pair
			pair0 = l_rotate_d(pair0 ^ l_rotate_d(pair0, 59) ^ l_rotate_d(pair0, 54), 40);
			pair1 = l_rotate_d(pair1 ^ l_rotate_d(pair1, 55) ^ l_rotate_d(pair1, 46), 32);
			pair2 = l_rotate_d(pair2 ^ l_rotate_d(pair2, 33) ^ l_rotate_d(pair2, 2), 16);
			pair3 = l_rotate_d(pair3 ^ l_rotate_d(pair3, 21) ^ l_rotate_d(pair3, 42), 56);
			pair4 = pair4 ^ l_rotate_d(pair4, 13) ^ l_rotate_d(pair4, 26);
		}

		// for the 4-round experiment (type-II), change it to:
		/*
		t1 = 0;
		t0 = initial0 & 0x66EEECDDD9BBB377; //2 round type-II distinguisher for 4 rounds
		for (int i = 0; i < 64; i++) t1 ^= ((t0 >> i) & 0x1);

		t0 = pair0 & 0x66EEECDDD9BBB377; //2 round type-II distinguisher for 4 round
		for (int i = 0; i < 64; i++) t1 ^= ((t0 >> i) & 0x1);

		if (t1 == 0) counter[threadIndex]++;
		*/

		// for the 5-round experiment, change it to (TYPE-I):

		t1 = 0;
		t0 = initial0 & 0x576eaedd55faabb7;
		t0 = t0 ^ (pair0 & 0x576eaedd55faabb7);
		for (int i = 0; i < 64; i++) t1 ^= (t0 >> i);

		t0 = initial1 & 0x6c1b168d8362d9b4;
		t0 = t0 ^ (pair1 & 0x6c1b168d8362d9b4);
		for (int i = 0; i < 64; i++) t1 ^= (t0 >> i);

		t0 = initial3 & 0x001ffffffffff800;
		t0 = t0 ^ (pair3 & 0x001ffffffffff800);
		for (int i = 0; i < 64; i++) t1 ^= (t0 >> i);

		if ((t1 & 0x1) == 0) counter[threadIndex]++;


		// for the 5-round experiment, change it to (TYPE-II):
		/*
		t1 = 0;
		t0 = initial0 & 0xB37766EECDDDD9BB; //2 round type-II distinguisher for 4 rounds
		for (int i = 0; i < 64; i++) t1 ^= ((t0 >> i) & 0x1);

		t0 = pair0 & 0xB37766EECDDDD9BB; //2 round type-II distinguisher for 4 round
		for (int i = 0; i < 64; i++) t1 ^= ((t0 >> i) & 0x1);

		if (t1 == 0) counter[threadIndex]++;
		*/
	}
}

__global__ void gpu_rotate(bit64 key[], bit64 nonce[], int key_choice, __int64 counter[], int rotation, int round) {
	// x0-x1: key, x2-x3: nonce, x4: IV
	// IV: Sycon-AEAD-96: 0x5980A92AFC5D9D2C
	// IV: Sycon-AEAD-64: 0x0000000000000000

	int threadIndex = blockIdx.x * blockDim.x + threadIdx.x;
	bit64 initial0, initial1, initial2, initial3, initial4 = 0x0000000000000000;
	bit64 pair0, pair1, pair2, pair3, pair4 = 0x0000000000000000;
	bit64 t0, t1, t2, t3, t4, t5, t6;

	initial2 = nonce[2 * threadIndex]; // nonce
	initial3 = nonce[2 * threadIndex + 1]; // nonce

	for (int c = 0; c < TRIALS; c++) {
		t0 = ((bit64)0x7FFFFFFFFFFFFFFF >> rotation) ^ ((bit64)0x7FFFFFFFFFFFFFFF << (64 - rotation));
		t1 = ((bit64)0x8000000000000000 >> rotation) ^ ((bit64)0x8000000000000000 << (64 - rotation));

		initial4 = 0x0000000000000000; // IV
		pair4 = 0x0000000000000000; // IV

		initial0 = key[0] & t0; if (key_choice == 2 || key_choice == 4) initial0 ^= t1; // key
		initial1 = key[1] & t0; if (key_choice == 3 || key_choice == 4) initial1 ^= t1; // key

		//1= (0, 0), 2= (0, 1), 3= (1, 0), 4= (1, 1)

		// the difference is on the nonce (x2-x3) for key recovery 
		/*
		pair0 = initial0;
		pair1 = initial1;
		pair2 = initial2 ^ t1;
		pair3 = initial3^ t1;
		*/

		// the difference is on the key (x0-x1) for related key attacks 
		/*
		pair0 = initial0^t1;
		pair1 = initial1^ t1;
		pair2 = initial2;
		pair3 = initial3;
		// pair4 = initial4;
		*/

		// the difference is on the nonce (x2-x3) and IV (x4) 

		pair0 = initial0;
		pair1 = initial1;
		pair2 = initial2 ^ t1;
		pair3 = initial3 ^ t1;
		pair4 = pair4 ^ t1;


		for (int i = 0; i < round; i++) {

			// Sycon Sbox // initial
			t0 = initial2 ^ initial4; t1 = t0 ^ initial1; t2 = initial1 ^ initial3; t3 = initial0 ^ initial4; t4 = t1 & initial3;
			t5 = t3 ^ t4; initial1 = ~initial1; initial1 = initial1 & initial3; t6 = ~t2; t6 = t6 & initial0;
			initial1 = initial1 ^ t1; initial1 = initial1 ^ t6; t3 = ~t3; t6 = t3 & initial2; t1 = t6 ^ t2;
			t0 = ~t0; initial3 = t0 & initial3; initial3 = initial3 ^ initial0; initial2 = ~initial2; initial3 = initial3 ^ initial2;
			initial4 = ~initial4; initial4 = initial4 & initial0; initial4 = initial4 ^ t2; initial0 = t5; initial2 = t1;

			// Sycon Liner layer // initial

			initial0 = l_rotate_d(initial0 ^ l_rotate_d(initial0, 59) ^ l_rotate_d(initial0, 54), 40);
			initial1 = l_rotate_d(initial1 ^ l_rotate_d(initial1, 55) ^ l_rotate_d(initial1, 46), 32);
			initial2 = l_rotate_d(initial2 ^ l_rotate_d(initial2, 33) ^ l_rotate_d(initial2, 2), 16);
			initial3 = l_rotate_d(initial3 ^ l_rotate_d(initial3, 21) ^ l_rotate_d(initial3, 42), 56);
			initial4 = initial4 ^ l_rotate_d(initial4, 13) ^ l_rotate_d(initial4, 26);
		}

		for (int i = 0; i < round; i++) {
			// Sycon sbox // pair
			t0 = pair2 ^ pair4; t1 = t0 ^ pair1; t2 = pair1 ^ pair3; t3 = pair0 ^ pair4; t4 = t1 & pair3;
			t5 = t3 ^ t4; pair1 = ~pair1; pair1 = pair1 & pair3; t6 = ~t2; t6 = t6 & pair0;
			pair1 = pair1 ^ t1; pair1 = pair1 ^ t6; t3 = ~t3; t6 = t3 & pair2; t1 = t6 ^ t2;
			t0 = ~t0; pair3 = t0 & pair3; pair3 = pair3 ^ pair0; pair2 = ~pair2; pair3 = pair3 ^ pair2;
			pair4 = ~pair4; pair4 = pair4 & pair0; pair4 = pair4 ^ t2; pair0 = t5; pair2 = t1;

			// Liner layer // pair
			pair0 = l_rotate_d(pair0 ^ l_rotate_d(pair0, 59) ^ l_rotate_d(pair0, 54), 40);
			pair1 = l_rotate_d(pair1 ^ l_rotate_d(pair1, 55) ^ l_rotate_d(pair1, 46), 32);
			pair2 = l_rotate_d(pair2 ^ l_rotate_d(pair2, 33) ^ l_rotate_d(pair2, 2), 16);
			pair3 = l_rotate_d(pair3 ^ l_rotate_d(pair3, 21) ^ l_rotate_d(pair3, 42), 56);
			pair4 = pair4 ^ l_rotate_d(pair4, 13) ^ l_rotate_d(pair4, 26);

		}
		// type-II - 3r
		t1 = 0;
		t0 = initial0 & 0xB37766EECDDDD9BB;
		t0 = t0 ^ (pair0 & 0xB37766EECDDDD9BB);
		for (int i = 0; i < 64; i++) t1 ^= (t0 >> i);
		if ((t1 & 0x1) == 0) counter[threadIndex]++;

		// type-II - 2r
		/*
		t1 = 0;
		t0 = initial0 & 0x66EEECDDD9BBB377;
		t0 = t0 ^ (pair0 & 0x66EEECDDD9BBB377);
		for (int i = 0; i < 64; i++) t1 ^= (t0 >> i);
		if ((t1 & 0x1) == 0) counter[threadIndex]++;
		*/

		initial2 += initial0;
		initial3 += initial1;
		// nonce    // key
	}
}

// #######################  SYCON EXPERIMENT - MAIN ######################

void show_menu() {
	printf(">>> SYCON Distinguisher Finder <<<\n\n"
		"(0) Test Vectors\n"
		"(1) CPU version\n"
		"(2) GPU version\n"
		"(3) GPU rotation version\n"
		"(4) Clear screen\n"
		"(5) Exit\n\n"
		"Choice: ");
}

void main() {
	int round = 0;
	bit64 experiment_size;
	int choice = 0;
	nonce = (bit64*)calloc(BLOCKS * THREADS * 2, sizeof(bit64));

	while (1) {

		show_menu();
		scanf_s("%d", &choice);

		if (choice == 0) { // (0) Test Vectors
			// to verify the test vectors
			test_permutation(test_state, 12);
		}

		if (choice == 1) { // (1) CPU version

			printf("Trial = 2^25 +  ");
			scanf_s("%I64d", &trial_i);
			trial = pow(2, trial_i);

			printf("For how many rounds: ");
			scanf_s("%d", &round);


			experiment_size = 1024 * 1024 * 32 * trial;
			printf("Running the experiment with %lld (2** %lld) data\n", experiment_size, trial_i + 25);
			bit64 ten_total = 0;
			for (int t = 0; t < repeat; t++) {
				ten_total += cpu_single_experiment(round, experiment_size);
			}
			printf("\nAverage bias: %lld\n", llabs(ten_total) / repeat);
		}

		if (choice == 2) { // (2) GPU version


			printf("Trial = 2^25 +  ");
			scanf_s("%I64d", &trial_i);
			trial = pow(2, trial_i);

			printf("For how many rounds: ");
			scanf_s("%d", &round);

			__int64* counter_d, * counter, total_counter = 0, bias, average_bias = 0;
			bit64 total_bias;
			// thread 32, block 1024, trials 1024 = 2**25 
			experiment_size = trial * TRIALS * THREADS * BLOCKS; // trial * 2**25
			printf("Running the experiment with %lld (2** %lld) data\n", experiment_size, trial_i + 25);

			for (int m = 0; m < repeat; m++) { // same experiment for "repeat" times
				counter = (__int64*)calloc(BLOCKS * THREADS, sizeof(bit64));
				total_counter = 0;
				hipMalloc((void**)&nonce_d, BLOCKS * THREADS * 2 * sizeof(bit64));
				hipMalloc((void**)&counter_d, BLOCKS * THREADS * sizeof(bit64));
				hipMemcpy(counter_d, counter, BLOCKS * THREADS * sizeof(__int64), hipMemcpyHostToDevice);


				// Create cuda events for measuring the time
				hipEvent_t start, stop;
				hipEventCreate(&start);
				hipEventCreate(&stop);

				// Start the timer
				hipEventRecord(start, 0);

				for (int i = 0; i < trial; i++) {
					for (int j = 0; j < THREADS * BLOCKS * 2; j++) { rdrand_64(nonce + j, 0); }
					hipMemcpy(counter, counter_d, BLOCKS * THREADS * sizeof(__int64), hipMemcpyDeviceToHost);
					hipMemcpy(counter_d, counter, BLOCKS * THREADS * sizeof(__int64), hipMemcpyHostToDevice);
					hipMemcpy(nonce_d, nonce, BLOCKS * THREADS * 2 * sizeof(bit64), hipMemcpyHostToDevice);
					gpu_single_experiment << <BLOCKS, THREADS >> > (nonce_d, counter_d, round);

				}

				// stop the timer
				hipEventRecord(stop, 0);
				hipEventSynchronize(stop);

				// calculate the elapsed time
				float elapsedTime;
				hipEventElapsedTime(&elapsedTime, start, stop);


				hipMemcpy(counter, counter_d, BLOCKS * THREADS * sizeof(__int64), hipMemcpyDeviceToHost);
				for (int i = 0; i < BLOCKS * THREADS; i++) total_counter += counter[i];
				bias = (experiment_size) / 2 - total_counter;
				printf("%03d: Total counter: %I64d Bias: %I64d Elapsed Time: %f second\n", m, total_counter, bias, elapsedTime / 1000.0f);
				total_bias = total_bias + llabs(bias);

				// destroy the events and free memory
				hipEventDestroy(start);
				hipEventDestroy(stop);
				hipFree(nonce_d); hipFree(counter_d);
			}
			printf("\nAverage Bias: %I64d \n", total_bias / repeat);

		}

		if (choice == 3) { // (3) GPU rotation version
			FILE* fp;
			int shift = 0, flag = 0, key_choice = 0;
			__int64* counter, * counter_d, total_counter = 0, bias, average_bias = 0, experiment;
			__int64 trial = 1, keys = 1; //10
			bit64 key[2], * key_d;
			printf("For how many rounds: ");
			scanf_s("%d", &round);

			printf("Pairs (2 ^ 25 + ?): ");
			scanf_s("%d", &shift);

			printf("Select key (1-4): ");
			scanf_s("%d", &key_choice);

			trial = 1;		trial = trial << shift;
			experiment = trial * TRIALS * THREADS * BLOCKS; // trial * 2**25
			if (key_choice == 1) fopen_s(&fp, "Automatic_search_key1.txt", "w");
			if (key_choice == 2) fopen_s(&fp, "Automatic_search_key2.txt", "w");
			if (key_choice == 3) fopen_s(&fp, "Automatic_search_key3.txt", "w");
			if (key_choice == 4) fopen_s(&fp, "Automatic_search_key4.txt", "w");
			printf("Key Choice: %d\n", key_choice); fprintf(fp, "Key Choice: %d\n", key_choice);
			printf("Pairs: 2 ^ %d\n", shift + 25); fprintf(fp, "Pairs: 2 ^ %d\n", shift + 25);
			printf("Experiment: %I64d\n", experiment); fprintf(fp, "Experiment: %I64d\n", experiment);
			for (int rotation = 0; rotation < 64; rotation++) {
				total_counter = 0; bias = 0; average_bias = 0; flag = 0;
				printf("Rotation: %d\n", rotation); fprintf(fp, "Rotation: %d\n", rotation);
				for (int m = 0; m < keys; m++) {
					counter = (__int64*)calloc(BLOCKS * THREADS, sizeof(bit64));
					total_counter = 0;
					hipMalloc((void**)&key_d, 2 * sizeof(bit64));
					hipMalloc((void**)&nonce_d, BLOCKS * THREADS * 2 * sizeof(bit64));
					hipMalloc((void**)&counter_d, BLOCKS * THREADS * sizeof(bit64));
					rdrand_64(key, 0);
					rdrand_64(key + 1, 0);
					StartCounter();
					hipMemcpy(counter_d, counter, BLOCKS * THREADS * sizeof(__int64), hipMemcpyHostToDevice);
					for (int i = 0; i < trial; i++) {
						for (int j = 0; j < THREADS * BLOCKS * 2; j++) { rdrand_64(nonce + j, 0); }
						hipMemcpy(counter, counter_d, BLOCKS * THREADS * sizeof(__int64), hipMemcpyDeviceToHost);
						hipMemcpy(counter_d, counter, BLOCKS * THREADS * sizeof(__int64), hipMemcpyHostToDevice);
						hipMemcpy(nonce_d, nonce, BLOCKS * THREADS * 2 * sizeof(bit64), hipMemcpyHostToDevice);
						hipMemcpy(key_d, key, 2 * sizeof(bit64), hipMemcpyHostToDevice);
						gpu_rotate << <BLOCKS, THREADS >> > (key_d, nonce_d, key_choice, counter_d, rotation, round);

					}
					hipMemcpy(counter, counter_d, BLOCKS * THREADS * sizeof(__int64), hipMemcpyDeviceToHost);
					for (int i = 0; i < BLOCKS * THREADS; i++) total_counter += counter[i];
					bias = (experiment) / 2 - total_counter;
					printf("%03d: Total counter: %I64d Bias: %I64d\n", m, total_counter, bias);
					fprintf(fp, "%03d: Total counter: %I64d Bias: %I64d\n", m, total_counter, bias);
					average_bias += bias;
					hipFree(key_d); hipFree(nonce_d); hipFree(counter_d);
					if (bias > 0 && flag < 0) m = keys + 1;
					else if (bias < 0 && flag > 0) m = keys + 1;
					if (bias > 0) flag = 1;
					else if (bias < 0) flag = -11;
				}
				average_bias /= keys;
				printf("Average bias: %I64d\n", llabs(average_bias)); fprintf(fp, "Average bias: %I64d\n", average_bias);
			}
			fclose(fp);
		}
		if (choice == 4) {
#ifdef _WIN32
			system("cls");
#else
			system("clear");
#endif
		}
		if (choice == 5) {
			printf("Exiting the program...\n");
			exit(0);
		}
	}
	system("PAUSE");
}